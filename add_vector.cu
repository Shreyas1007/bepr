
#include "hip/hip_runtime.h"


#include <stdio.h>
#define N 2048
#define THREADS_PER_BLOCK 50

__global__ void add(int *a, int *b, int *c)
{

	printf("threadid No : %d\n",threadIdx.x);
	printf("blockid No : %d\n",blockIdx.x);
	printf("blockdim No : %d\n",blockDim.x);

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	

	printf("Index No : %d\n",index);

	c[index] = a[index] + b[index];

}

int main()
{
	int a[N],b[N],c[N];
	int *dev_a, *dev_b, *dev_c;
	int size = N * sizeof(int);


	hipMalloc((void **)&dev_a,size);
	hipMalloc((void **)&dev_b,size);
	hipMalloc((void **)&dev_c,size);

	//FILL data
	for(int i=0;i<10;i++)
	{
		a[i]=i;
		b[i]=i;
	}

	hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

	for(int i=0;i<10;i++)
	{
		printf("\n %d + %d --> addition is :%d \n\n",a[i],b[i],c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;

}