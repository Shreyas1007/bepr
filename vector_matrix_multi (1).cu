#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>

using namespace std;
#define N 3
#define NN 50
#define BLOCKSIZE 10
#define __HIPCC__
#include <hip/device_functions.h>


__global__ void compute(int *a, int *b, int *c, int rowsize)
{
		int tidx = blockIdx.x*blockDim.x + threadIdx.x;
        int tidy = blockIdx.y*blockDim.y + threadIdx.y;
        int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;
		 if(tindex<rowsize)
		 {
			 int i;int m=tindex*rowsize;
				c[tindex]=0.00;
			
				for(i=0;i<N;i++)
			  c[tindex]+=a[m+i]*b[i];
		}

    __syncthreads();
			cout<<"\n tidx :"<<tidx;
			cout<<"\n tidy :"<<tidy;
			cout<<"\n tindex :"<<tindex;

}

int main()
{
	int vector[N],matrix[N][N],result[N];
	int *dev_vector, *dev_matrix, *dev_c;
	int size = NN*sizeof(int);

	cout<<"\n Enter 3 element :";
	for(int i=0;i<3;i++)
	{
		cin>>vector[i];		
	}
	cout<<"\n Enter 3x3 matrix element:";
	for(int i=0;i<3;i++)
	{
		for(int j=0;j<3;j++)
		{
				cin>>matrix[i][j];
		}
	}

	cout<<"\n Entered element:";
	for(int i=0;i<3;i++)
	{    
		cout<<"\n";
		for(int j=0;j<3;j++)
		{
				cout<<matrix[i][j];
				cout<<"\t";
		}
		cout<<"\t \t";
	}
	hipMalloc((void **)&dev_vector,size);
	hipMalloc((void **)&dev_matrix,size);
	hipMalloc((void **)&dev_c,size);

	hipMemcpy(dev_vector,vector,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_matrix,matrix,size,hipMemcpyHostToDevice);

	compute<<<N,N>>>(dev_vector,dev_matrix,dev_c,N);

	hipMemcpy(result,dev_c,size,hipMemcpyDeviceToHost);
	
	for(int j=0;j<3;j++)
	{
		cout<<"result"<<result[j];
		cout<<"\t";
	}

	return 0;
}