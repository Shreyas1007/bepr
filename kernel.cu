#include "hip/hip_runtime.h"


#include <ctime>
#include <stdio.h>
#include <iostream>
#include <math.h>

using namespace std;


__global__ void MulKernel(int *c, const int *a, const int *b, const int P)
{
    int tempsum=0;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    if (row < P && col < P){
        for (int i = 0; i < P; i++){
            tempsum += a[row*P + i] * b[i*P + col];
        }
        c[row*P + col] = tempsum;
    }
}


int main()
{

    srand(time(NULL));
    int N = 16;
    int SIZE = N*N;

    int *h_a = new int[SIZE];
    int *h_b = new int[SIZE];
    int *h_c = new int[SIZE];

    for (int i = 0; i < SIZE; i++) {
            h_a[i] = rand() % 1000;
            h_b[i] = rand() % 1000;
    }
    cout << "First values " << h_a[0] << " " << h_b[0] << endl;
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(int)*SIZE);
    hipMalloc(&d_b, sizeof(int)*SIZE);
    hipMalloc(&d_c, sizeof(int)*SIZE);

    cout << "Second values " << h_a[0] << " " << h_b[0] << endl;

    hipMemcpy(d_a, h_a, sizeof(int)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*SIZE, hipMemcpyHostToDevice);

    cout << "Third values " << h_a[0] <<" "<< h_b[0] << endl;

    MulKernel <<<1, dim3(N,N) >>>(d_c, d_a, d_b, N);

    hipMemcpy(h_c, d_c, sizeof(int)*SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(h_a, d_a, sizeof(int)*SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, sizeof(int)*SIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < 5; i++){
        cout << h_c[i] << "=" << h_a[i] << h_b[i] << endl;
    }
    cout << h_c[1] << endl;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}