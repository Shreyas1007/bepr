#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
#include<math.h>
using namespace std;
#define N 20

__global__ void max(int *a, int *b)
{
	*b=a[0];

	if(*a > *b)
	{
		*b = *a;
	}

}

__global__ void min(int *a, int *b)
{
	*b=a[0];

	if(*a < *b)
	{
		*b = *a;
	}

}

__global__ void avg(int *a,int *b)
{
	*b = *b + *a;
	
}


int main() 
{ 
	int ch,size=0,a[N],max,min,avg=0;
	char ans;
	float sum = 0.0, mean, standardDeviation = 0.0,varsum=0.0;
	    
	int *dev_a,*dev_b; //GPU
	int size = N*sizeof(int);

	cout<<"\n Enter Size of Vector";
	cin>>size;
	cout<<"\n Enter Element :";
	for(int i=0;i<size;i++)
	{

		cin>>a[i];
	}
	
		hipMalloc((void **)&dev_a,size);
		hipMalloc((void **)&dev_b,size);
		dev_b=&a[0];
		hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
									

	do
	{
		cout<<"\n -----MENU-----";
		cout<<"\n 1.Find Max number from vector";
		cout<<"\n 2.Find Min number from vector";
		cout<<"\n 3.Arithmetic Mean ";
		cout<<"\n 4.Standard Deviation";
		cout<<"\n 5.exit";
		cout<<"\n Enter ur choice:";
		cin>>ch;
		switch(ch)
		{

			case 1:
						
				    //------------------------GPU CODE--------------

					max<<<1,1>>>(dev_a,dev_b);

					hipMemcpy(&max,dev_b,size,hipMemcpyDeviceToHost);

					//----------------------------------------------
					cout<<"\n Max element in Vector is :"<<max<<"\n";

					break;
			case 2:
					 //------------------------GPU CODE--------------

					min<<<1,1>>>(dev_a,dev_b);

					hipMemcpy(&min,dev_b,size,hipMemcpyDeviceToHost);

					//----------------------------------------------
					cout<<"\n Min element in Vector is :"<<min<<"\n";
					break;
			case 3:
					

					avg<<<1,1>>>(dev_a,dev_b);

					hipMemcpy(&sum,dev_b,size,hipMemcpyDeviceToHost);

					avg = sum/size;

					cout<<"\n Arithmetic Mean is :"<<avg<<"\n";
					break;
			case 4:  //summation (vector - (Mean))sqr/n-1
					
				standardDeviation=0.0;
				sum=0;

					for(int i=0; i<size; i++)
					{
						sum	= sum + a[i];
					}
					mean = sum/size;
				
					for(int i=0; i<size; i++)
					{
						float  diff = a[i] - mean;
						 varsum = varsum + pow(diff,2);
					}

					float  variance = varsum/size;

					cout<<"\n Variance is :"<<variance;

					standardDeviation = sqrt(variance);
					
					cout<<"\n Std Deviation is :"<<standardDeviation<<"\n"; 

					break;

			case 5:
					cout<<"\n Thanks\n\n\n";
					exit(0);
					break;
		}
		cout<<"\n Do u want to continue ?";
		cin>>ans;
	}while(ans=='y'||ans=='Y');


    return 0; 
} 